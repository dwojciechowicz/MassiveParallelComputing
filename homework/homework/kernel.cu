#include "hip/hip_runtime.h"
#include <cstdio>
#include <Windows.h>
#include <WinBase.h>
#include <stdlib.h>

#if defined(NDEBUG)
#define CUDA_CHECK(x)	(x)
#else
#define CUDA_CHECK(x)	do {\
		(x); \
		hipError_t e = hipGetLastError(); \
		if (hipSuccess != e) { \
			printf("cuda failure \"%s\" at %s:%d\n", \
			       hipGetErrorString(e), \
			       __FILE__, __LINE__); \
			exit(1); \
		} \
	} while (0)
#endif
//data generator
void generateData(float* ptr, unsigned int size) {
	while (size--) {
		*ptr++ = (float)(rand() % 1000) / 1000.0F;
	}
}

// kernel program
__global__ void mulKernel(float* p, const float* m, const float* n, const int WIDTH) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int i = y * WIDTH + x;
	float sum = 0.0f;
	for (int k = 0; k < WIDTH; ++k) {
		sum += m[y * WIDTH + k] * n[k * WIDTH + x];
	}
	p[i] = sum;
}


int main(void) {
	// host-side data
	long long start, end, f;
	QueryPerformanceFrequency((LARGE_INTEGER*)(&f));
	const int WIDTH = 4096;
	const int TILE_WIDTH = 32;
	float* M=NULL;
	float* N=NULL;
	//float* P=NULL;
	float (*P)[WIDTH];

	M = (float*)malloc(WIDTH * WIDTH * sizeof(float));
	N = (float*)malloc(WIDTH * WIDTH * sizeof(float));
	P = (float(*)[WIDTH])malloc(WIDTH * WIDTH * sizeof(float));
	//P = (float*)malloc(WIDTH * WIDTH * sizeof(float));

	generateData(M, WIDTH * WIDTH);
	generateData(N, WIDTH * WIDTH);

	// device-side data
	float* dev_m = 0;
	float* dev_n = 0;
	float* dev_p = 0;
	
	CUDA_CHECK(hipMalloc((void**)&dev_m, WIDTH * WIDTH * sizeof(float)));
	CUDA_CHECK(hipMalloc((void**)&dev_n, WIDTH * WIDTH * sizeof(float)));
	CUDA_CHECK(hipMalloc((void**)&dev_p, WIDTH * WIDTH * sizeof(float)));
	CUDA_CHECK(hipMemcpy(dev_m, M, WIDTH * WIDTH * sizeof(float), hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(dev_n, N, WIDTH * WIDTH * sizeof(float), hipMemcpyHostToDevice));
	
	dim3 dimGrid(WIDTH/TILE_WIDTH, WIDTH/TILE_WIDTH, 1);
	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
	QueryPerformanceCounter((LARGE_INTEGER*)(&start));//I am interested only in the time of CUDA kernel function execution
	mulKernel <<< dimGrid, dimBlock >>> (dev_p, dev_m, dev_n, WIDTH);
	QueryPerformanceCounter((LARGE_INTEGER*)(&end));
	CUDA_CHECK(hipPeekAtLastError());
	CUDA_CHECK(hipMemcpy(P, dev_p, WIDTH * WIDTH * sizeof(float), hipMemcpyDeviceToHost));
	CUDA_CHECK(hipFree(dev_m));
	CUDA_CHECK(hipFree(dev_n));
	CUDA_CHECK(hipFree(dev_p));

	//printf("%5f\n", P[0]);
	//printf("%5f\n", P[123 * WIDTH + 456]);
	//printf("%5f\n", P[WIDTH * WIDTH - 1]);
	printf("%5f\n", P[0][0]);
	printf("%5f\n", P[123][456]);
	printf("%5f\n", P[WIDTH-1][WIDTH-1]);
	printf("\nelapsed time = %f usec\n", (double)(end - start) * 1000000.0 / (double)(f));
	free(M);
	free(N);
	free(P);
	return 0;
}

